#include "hip/hip_runtime.h"
#include "gpurasteriser.cuh"
#include "utilities/OBJLoader.hpp"
#include <vector>
#include <iomanip>
#include <chrono>
#include <limits>
#include <iostream>
#include <algorithm>
#include "hip/hip_runtime.h"
#include "utilities/cuda_error_helper.hpp"


// UTILITY FUNCTIONS HAVE BEEN MOVED INTO THE KERNEL SOURCE FILE ITSELF
// CUDA relocatable and separable compilation is possible, but due to the many possible
// problems it can cause on different platforms, I decided to take the safe route instead
// and make sure it would compile fine for everyone. That implies moving everything into
// one file unfortunately.

class globalLight {
public:
	float3 direction;
	float3 colour;
	__host__ __device__ globalLight(float3 const vdirection, float3 const vcolour) : direction(vdirection), colour(vcolour) {}
};

__host__ __device__ float dotGPU(float3 a, float3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ float3 normalizeGPU(float3 v)
{
    float invLen = 1.0f / sqrtf(dotGPU(v, v));
    v.x *= invLen;
    v.y *= invLen;
    v.z *= invLen;
    return v;
}

// Utility function if you'd like to convert the depth buffer to an integer format.
__host__ __device__ int depthFloatToInt(float value) {
	value = (value + 1.0f) * 0.5f;
    return static_cast<int>(static_cast<double>(value) * static_cast<double>(16777216)); 
}

__host__ __device__ bool isPointInTriangle(
		float4 const &v0, float4 const &v1, float4 const &v2,
		unsigned int const x, unsigned int const y,
		float &u, float &v, float &w) {
		u = (((v1.y - v2.y) * (x    - v2.x)) + ((v2.x - v1.x) * (y    - v2.y))) /
				 	 (((v1.y - v2.y) * (v0.x - v2.x)) + ((v2.x - v1.x) * (v0.y - v2.y)));
		if (u < 0) {
			return false;
		}
		v = (((v2.y - v0.y) * (x    - v2.x)) + ((v0.x - v2.x) * (y    - v2.y))) /
					(((v1.y - v2.y) * (v0.x - v2.x)) + ((v2.x - v1.x) * (v0.y - v2.y)));
		if (v < 0) {
			return false;
		}
		w = 1 - u - v;
		if (w < 0) {
			return false;
		}
		return true;
}

__host__ __device__ float3 computeInterpolatedNormal(
		float3 const &normal0,
		float3 const &normal1,
		float3 const &normal2,
		float3 const &weights
	) {
	float3 weightedN0, weightedN1, weightedN2;

	weightedN0.x = (normal0.x * weights.x);
	weightedN0.y = (normal0.y * weights.x);
	weightedN0.z = (normal0.z * weights.x);

	weightedN1.x = (normal1.x * weights.y);
	weightedN1.y = (normal1.y * weights.y);
	weightedN1.z = (normal1.z * weights.y);

	weightedN2.x = (normal2.x * weights.z);
	weightedN2.y = (normal2.y * weights.z);
	weightedN2.z = (normal2.z * weights.z);

	float3 weightedNormal;

	weightedNormal.x = weightedN0.x + weightedN1.x + weightedN2.x;
	weightedNormal.y = weightedN0.y + weightedN1.y + weightedN2.y;
	weightedNormal.z = weightedN0.z + weightedN1.z + weightedN2.z;

	return normalizeGPU(weightedNormal);
}

__host__ __device__ float computeDepth(
		float4 const &v0, float4 const &v1, float4 const &v2,
		float3 const &weights) {
	return weights.x * v0.z + weights.y * v1.z + weights.z * v2.z;
}





// ORIGINAL SOURCE FILE IS STARTING HERE

struct workItemGPU {
    float scale;
    float3 distanceOffset;

    workItemGPU(float& scale_, float3& distanceOffset_) : scale(scale_), distanceOffset(distanceOffset_) {}
    workItemGPU() : scale(1), distanceOffset(make_float3(0, 0, 0)) {}
};

void runVertexShader( float4 &vertex,
                      float3 positionOffset,
                      float scale,
					  unsigned int const width,
					  unsigned int const height,
				  	  float const rotationAngle = 0)
{
	float const pi = 3.1415926f;
	// The matrices defined below are the ones used to transform the vertices and normals.

	// This projection matrix assumes a 16:9 aspect ratio, and an field of view (FOV) of 90 degrees.
	mat4x4 const projectionMatrix(
		0.347270,   0, 			0, 		0,
		0,	  		0.617370, 	0,		0,
		0,	  		0,			-1, 	-0.2f,
		0,	  		0,			-1,		0);

	mat4x4 translationMatrix(
		1,			0,			0,			0 + positionOffset.x /*X*/,
		0,			1,			0,			0 + positionOffset.y /*Y*/,
		0,			0,			1,			-10 + positionOffset.z /*Z*/,
		0,			0,			0,			1);

	mat4x4 scaleMatrix(
		scale/*X*/,	0,			0,				0,
		0, 			scale/*Y*/, 0,				0,
		0, 			0,			scale/*Z*/, 	0,
		0, 			0,			0,				1);

	mat4x4 const rotationMatrixX(
		1,			0,				0, 				0,
		0, 			cosf(0), 	-sinf(0),	0,
		0, 			sinf(0),	cosf(0), 	0,
		0, 			0,				0,				1);

	float const rotationAngleRad = (pi / 4.0f) + (rotationAngle / (180.0f/pi));

	mat4x4 const rotationMatrixY(
		cosf(rotationAngleRad), 0, sinf(rotationAngleRad), 0,
		0, 1, 0, 0,
		-sinf(rotationAngleRad), 0, cosf(rotationAngleRad), 	0,
		0, 0, 0, 1);

	mat4x4 const rotationMatrixZ(
		cosf(pi),	-sinf(pi),	0,			0,
		sinf(pi), 	cosf(pi), 	0,			0,
		0,				0,				1,			0,
		0, 				0,				0,			1);

	mat4x4 const MVP =
		projectionMatrix * translationMatrix * rotationMatrixX * rotationMatrixY * rotationMatrixZ * scaleMatrix;

		float4 transformed = (MVP * vertex);

    vertex.x = transformed.x / transformed.w;
    vertex.y = transformed.y / transformed.w;
    vertex.z = transformed.z / transformed.w;
    vertex.w = 1.0;

    vertex.x = (vertex.x + 0.5f) * (float) width;
    vertex.y = (vertex.y + 0.5f) * (float) height;
}


void runFragmentShader( unsigned char* frameBuffer,
						unsigned int const baseIndex,
						GPUMesh &mesh,
						unsigned int triangleIndex,
						float3 const &weights)
{
	float3 normal = computeInterpolatedNormal(
            mesh.normals[3 * triangleIndex + 0],
            mesh.normals[3 * triangleIndex + 1],
            mesh.normals[3 * triangleIndex + 2],
			weights);

    float3 colour = make_float3(0.0f, 0.0f, 0.0f);

    const unsigned int lightSourceCount = 1;
    const globalLight lightSources[lightSourceCount] = {{make_float3(0.3f, 0.5f, 1.0f), make_float3(1.0f, 1.0f, 1.0f)}};

	for (unsigned int lightSource = 0; lightSource < lightSourceCount; lightSource++) {
		globalLight l = lightSources[lightSource];
		float lightNormalDotProduct = 
			normal.x * l.direction.x + normal.y * l.direction.y + normal.z * l.direction.z;

		float3 diffuseReflectionColour;
		diffuseReflectionColour.x = mesh.objectDiffuseColour.x * l.colour.x;
		diffuseReflectionColour.y = mesh.objectDiffuseColour.y * l.colour.y;
		diffuseReflectionColour.z = mesh.objectDiffuseColour.z * l.colour.z;

		colour.x += diffuseReflectionColour.x * lightNormalDotProduct;
		colour.y += diffuseReflectionColour.y * lightNormalDotProduct;
		colour.z += diffuseReflectionColour.z * lightNormalDotProduct;
	}

    colour.x = fminf(fmaxf(colour.x, 0.0f), 1.0f);
    colour.y = fminf(fmaxf(colour.y, 0.0f), 1.0f);
    colour.z = fminf(fmaxf(colour.z, 0.0f), 1.0f);

    frameBuffer[4 * baseIndex + 0] = colour.x * 255.0f;
    frameBuffer[4 * baseIndex + 1] = colour.y * 255.0f;
    frameBuffer[4 * baseIndex + 2] = colour.z * 255.0f;
    frameBuffer[4 * baseIndex + 3] = 255;

}

/**
 * The main procedure which rasterises all triangles on the framebuffer
 * @param transformedMesh         Transformed mesh object
 * @param frameBuffer             frame buffer for the rendered image
 * @param depthBuffer             depth buffer for every pixel on the image
 * @param width                   width of the image
 * @param height                  height of the image
 */
void rasteriseTriangle( float4 &v0, float4 &v1, float4 &v2,
                        GPUMesh &mesh,
                        unsigned int triangleIndex,
                        unsigned char* frameBuffer,
                        int* depthBuffer,
                        unsigned int const width,
                        unsigned int const height ) {

    // Compute the bounding box of the triangle.
    // Pixels that are intersecting with the triangle can only lie in this rectangle
	unsigned int minx = unsigned(floorf(fminf(fminf(v0.x, v1.x), v2.x)));
	unsigned int maxx = unsigned(ceilf(fmaxf(fmaxf(v0.x, v1.x), v2.x)));
	unsigned int miny = unsigned(floorf(fminf(fminf(v0.y, v1.y), v2.y)));
	unsigned int maxy = unsigned(ceilf(fmaxf(fmaxf(v0.y, v1.y), v2.y)));

	// Make sure the screen coordinates stay inside the window
    // This ensures parts of the triangle that are outside the
    // view of the camera are not drawn.
	minx = fmaxf(minx, (unsigned int) 0);
	maxx = fminf(maxx, width);
	miny = fmaxf(miny, (unsigned int) 0);
	maxy = fminf(maxy, height);

	// We iterate over each pixel in the triangle's bounding box
	for (unsigned int x = minx; x < maxx; x++) {
		for (unsigned int y = miny; y < maxy; y++) {
			float u, v, w;
			// For each point in the bounding box, determine whether that point lies inside the triangle
			if (isPointInTriangle(v0, v1, v2, x, y, u, v, w)) {
				// If it does, compute the distance between that point on the triangle and the screen
				float pixelDepth = computeDepth(v0, v1, v2, make_float3(u, v, w));
				// If the point is closer than any point we have seen thus far, render it.
				// Otherwise it is hidden behind another object, and we can throw it away
				// Because it will be invisible anyway.
                if (pixelDepth >= -1 && pixelDepth <= 1) {
					int pixelDepthConverted = depthFloatToInt(pixelDepth);
                 	if (pixelDepthConverted < depthBuffer[y * width + x]) {
					    // If it is, we update the depth buffer to the new depth.
					    depthBuffer[y * width + x] = pixelDepthConverted;

					    // And finally we determine the colour of the pixel, now that 
					    // we know our pixel is the closest we have seen thus far.
						runFragmentShader(frameBuffer, x + (width * y), mesh, triangleIndex, make_float3(u, v, w));
					}
				}
			}
		}
	}
}


void renderMeshes(
        unsigned long totalItemsToRender,
        workItemGPU* workQueue,
        GPUMesh* meshes,
        unsigned int meshCount,
        unsigned int width,
        unsigned int height,
        unsigned char* frameBuffer,
        int* depthBuffer
) {
    int count = 0;
    for(unsigned int item = 0; item < totalItemsToRender; item++) {
        for (unsigned int meshIndex = 0; meshIndex < meshCount; meshIndex++) {
            for(unsigned int triangleIndex = 0; triangleIndex < meshes[meshIndex].vertexCount / 3; triangleIndex++) {
            	count++;
	    }
        }
    }
    std::cout << "The triple loop runs " << count << " times." << std::endl;

    double avg1 = 0; double avg2 = 0; double avg3 = 0;
    int count1 = 0; int count2 = 0; int count3 = 0;

    for(unsigned int item = 0; item < totalItemsToRender; item++) {
        auto start1 = std::chrono::high_resolution_clock::now();
        workItemGPU objectToRender = workQueue[item];
        for (unsigned int meshIndex = 0; meshIndex < meshCount; meshIndex++) {
            auto start2 = std::chrono::high_resolution_clock::now();
            for(unsigned int triangleIndex = 0; triangleIndex < meshes[meshIndex].vertexCount / 3; triangleIndex++) {
                auto start3 = std::chrono::high_resolution_clock::now();

		float4 v0 = meshes[meshIndex].vertices[triangleIndex * 3 + 0];
                float4 v1 = meshes[meshIndex].vertices[triangleIndex * 3 + 1];
                float4 v2 = meshes[meshIndex].vertices[triangleIndex * 3 + 2];

                runVertexShader(v0, objectToRender.distanceOffset, objectToRender.scale, width, height);
                runVertexShader(v1, objectToRender.distanceOffset, objectToRender.scale, width, height);
                runVertexShader(v2, objectToRender.distanceOffset, objectToRender.scale, width, height);

                rasteriseTriangle(v0, v1, v2, meshes[meshIndex], triangleIndex, frameBuffer, depthBuffer, width, height);
            	
		auto end3 = std::chrono::high_resolution_clock::now();
    		std::chrono::duration<double> time3 = end3 - start3;
		avg3 += time3.count(); count3++;
	    }
            auto end2 = std::chrono::high_resolution_clock::now();
    	    std::chrono::duration<double> time2 = end2 - start2;
	    avg2 += time2.count(); count2++;
        }
        auto end1 = std::chrono::high_resolution_clock::now();
    	std::chrono::duration<double> time1 = end1 - start1;
	avg1 += time1.count(); count1++;
    }
    std::cout << "The average time it took to run a first loop is " << avg1 / count1 << " seconds." << std::endl;
    std::cout << "The average time it took to run a second loop is " << avg2 / count2 << " seconds." << std::endl;
    std::cout << "The average time it took to run a third loop is " << avg3 / count3 << " seconds." << std::endl;
}



void fillWorkQueue(
        workItemGPU* workQueue,
        float largestBoundingBoxSide,
        int depthLimit,
        unsigned long* nextIndexInQueue,
        float scale = 1.0,
        float3 distanceOffset = {0, 0, 0}) {

    // Queue a work item at the current scale and location
    workQueue[*nextIndexInQueue] = {scale, distanceOffset};
    (*nextIndexInQueue)++;

    // Check whether we've reached the recursive depth of the fractal we want to reach
    depthLimit--;
    if(depthLimit == 0) {
        return;
    }

    // Now we recursively draw the meshes in a smaller size
    for(int offsetX = -1; offsetX <= 1; offsetX++) {
        for(int offsetY = -1; offsetY <= 1; offsetY++) {
            for(int offsetZ = -1; offsetZ <= 1; offsetZ++) {
                float3 offset = make_float3(offsetX,offsetY,offsetZ);
                // We draw the new objects in a grid around the "main" one.
                // We thus skip the location of the object itself.
                if(offsetX == 0 && offsetY == 0 && offsetZ == 0) {
                    continue;
                }

                float smallerScale = scale / 3.0f;
                float3 displacedOffset = make_float3(
                        distanceOffset.x + offset.x * (largestBoundingBoxSide / 2.0f) * scale,
                        distanceOffset.y + offset.y * (largestBoundingBoxSide / 2.0f) * scale,
                        distanceOffset.z + offset.z * (largestBoundingBoxSide / 2.0f) * scale
                );

                fillWorkQueue(workQueue, largestBoundingBoxSide, depthLimit, nextIndexInQueue, smallerScale, displacedOffset);
            }
        }
    }

}


__global__ void initializeFrameBuffer(unsigned char* framePointer) {
	int xThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yThreadIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int threadIndex = yThreadIndex * gridDim.x * blockDim.x + xThreadIndex;
	if (threadIndex % 4 == 0) {
		framePointer[threadIndex] = 255;
	} else {
		framePointer[threadIndex] = 0;
	}
	//printf("I'm thread number %i, and I just assigned the value %i to the frame buffer!\n", threadIndex, framePointer[threadIndex]);

}

__global__ void initializeDepthBuffer(int* depthPointer) {
	int xThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yThreadIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int threadIndex = yThreadIndex * gridDim.x * blockDim.x + xThreadIndex;
	depthPointer[threadIndex] = 16777216;
	//printf("I'm thread number %i, and I just assigned the value %i to the depth buffer!\n", threadIndex, depthPointer[threadIndex]);
}




// This function kicks off the rasterisation process.
std::vector<unsigned char> rasteriseGPU(std::string inputFile, unsigned int width, unsigned int height, unsigned int depthLimit) {
    std::cout << "Rendering an image on the GPU.." << std::endl;
    std::cout << "Loading '" << inputFile << "' file... " << std::endl;

    std::vector<GPUMesh> meshes = loadWavefrontGPU(inputFile, false);

    int count = 0;
    checkCudaErrors(hipGetDeviceCount(&count));
    std::cout << "There are " << count << " GPU devices on the CUDA driver." << std::endl;

    hipDeviceProp_t* prop = new hipDeviceProp_t;
    int device = 0;
    checkCudaErrors(hipGetDeviceProperties(prop, device));
    std::cout << "The name of this CUDA device is " << prop->name << std::endl;

    checkCudaErrors(hipSetDevice(0));

    // We first need to allocate some buffers.
    // The framebuffer contains the image being rendered.
    unsigned char* frameBuffer = new unsigned char[width * height * 4];

    // The depth buffer is used to make sure that objects closer to the camera occlude/obscure objects that are behind it
    for (unsigned int i = 0; i < (4 * width * height); i+=4) {
		frameBuffer[i + 0] = 0;
		frameBuffer[i + 1] = 0;
		frameBuffer[i + 2] = 0;
		frameBuffer[i + 3] = 255;
	}

	int* depthBuffer = new int[width * height];
	for(unsigned int i = 0; i < width * height; i++) {
    	depthBuffer[i] = 16777216; // = 2 ^ 24
    }
    
    //We need to allocate more buffers on the GPU for both frame and depth.
    size_t frameSizeInBytes = width * height * 4 * sizeof(unsigned char);		
    size_t depthSizeInBytes = width * height * sizeof(int);		
    unsigned char* framePointer = 0;
    int* depthPointer = 0;
    checkCudaErrors(hipMalloc(&framePointer, frameSizeInBytes));
    checkCudaErrors(hipMalloc(&depthPointer, depthSizeInBytes));

    //checkCudaErrors(hipMemcpy(framePointer, frameBuffer, frameSizeInBytes, hipMemcpyHostToDevice));
    //checkCudaErrors(hipMemcpy(depthPointer, depthBuffer, depthSizeInBytes, hipMemcpyHostToDevice));

    int frameGridSize = (width * height * 4 + 127) / 128;
    int depthGridSize = (width * height + 31) / 32;
    std::cout << "Maximum frame grid size is " << frameGridSize << std::endl;
    std::cout << "Maximum depth grid size is " << depthGridSize << std::endl;
    dim3 frameBlock(32, 4, 1);
    dim3 frameGrid(frameGridSize, 1, 1);
    dim3 depthBlock(32, 1, 1);
    dim3 depthGrid(depthGridSize, 1, 1);
    initializeFrameBuffer<<<frameGrid, frameBlock>>>(framePointer);
    initializeDepthBuffer<<<depthGrid, depthBlock>>>(depthPointer);

    checkCudaErrors(hipDeviceSynchronize());

    float3 boundingBoxMin = make_float3(std::numeric_limits<float>::max(), std::numeric_limits<float>::max(), std::numeric_limits<float>::max());
    float3 boundingBoxMax = make_float3(std::numeric_limits<float>::min(), std::numeric_limits<float>::min(), std::numeric_limits<float>::min());

    std::cout << "Rendering image... " << std::endl;

    for(unsigned int i = 0; i < meshes.size(); i++) {
        for(unsigned int vertex = 0; vertex < meshes.at(i).vertexCount; vertex++) {
            boundingBoxMin.x = std::min(boundingBoxMin.x, meshes.at(i).vertices[vertex].x);
            boundingBoxMin.y = std::min(boundingBoxMin.y, meshes.at(i).vertices[vertex].y);
            boundingBoxMin.z = std::min(boundingBoxMin.z, meshes.at(i).vertices[vertex].z);

            boundingBoxMax.x = std::max(boundingBoxMax.x, meshes.at(i).vertices[vertex].x);
            boundingBoxMax.y = std::max(boundingBoxMax.y, meshes.at(i).vertices[vertex].y);
            boundingBoxMax.z = std::max(boundingBoxMax.z, meshes.at(i).vertices[vertex].z);
        }
    }

    float3 boundingBoxDimensions = make_float3(
            boundingBoxMax.x - boundingBoxMin.x,
            boundingBoxMax.y - boundingBoxMin.y,
            boundingBoxMax.z - boundingBoxMin.z);
    float largestBoundingBoxSide = std::max(std::max(boundingBoxDimensions.x, boundingBoxDimensions.y), boundingBoxDimensions.z);

    // Each recursion level splits up the lowest level nodes into 28 smaller ones.
    // This regularity means we can calculate the total number of objects we need to render
    // which we can of course preallocate
    unsigned long totalItemsToRender = 0;
    for(unsigned long level = 0; level < depthLimit; level++) {
        totalItemsToRender += std::pow(26ul, level);
    }

    workItemGPU* workQueue = new workItemGPU[totalItemsToRender];

    std::cout << "Number of items to be rendered: " << totalItemsToRender << std::endl;

    auto start = std::chrono::high_resolution_clock::now();
    
    unsigned long counter = 0;
    fillWorkQueue(workQueue, largestBoundingBoxSide, depthLimit, &counter);

	renderMeshes(
			totalItemsToRender, workQueue,
			meshes.data(), meshes.size(),
			width, height, frameBuffer, depthBuffer);
    
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> time_taken = end - start;

    std::cout << "The time it took to run the work queue is " << time_taken.count() << " seconds." << std::endl;

    std::cout << "Finished!" << std::endl;

    delete prop; delete frameBuffer; delete depthBuffer;
    hipFree(framePointer);
    hipFree(depthPointer);

    // Copy the output picture into a vector so that the image dump code is happy :)
    std::vector<unsigned char> outputFramebuffer(frameBuffer, frameBuffer + (width * height * 4));

    return outputFramebuffer;
}
